#include "hip/hip_runtime.h"
#include "GpuProjectionProcessing.cuh"
#include "GpuUtmWgsTransform.cuh"
#include "GpuTimer.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

using namespace winGpu;

__global__ void applyTransformUtmToWgsCoordsGpu(double xOrigin, double yOrigin, double xPixelSize, double yPixelSize,
	int height, int width, int zone, bool southhemi, double* lon, double* lat)
{
	int h = blockDim.x * blockIdx.x + threadIdx.x;
	int w = blockDim.y * blockIdx.y + threadIdx.y;
	if (height <= h || width <= w)
	{
		return;
	}
	double x = xOrigin + xPixelSize * w;
	double y = yOrigin + yPixelSize * h;
	double newLon = 0.0;
	double newLat = 0.0;

	UtmXYToLatLonGpu(x, y, zone, southhemi, newLon, newLat);

	lon[h * width + w] = newLon;
	lat[h * width + w] = newLat;
}

double winGpu::doTransformUtmToWgsCoordsGpu(double xOrigin, double yOrigin, double xPixelSize, double yPixelSize,
	int height, int width, int zone, bool southhemi, double* lon, double* lat)
{
	const size_t maxAvaliableCoords = 2000000;
	int countRowsPerIter = maxAvaliableCoords / width;
	int countIter = height / countRowsPerIter + 1;
	const size_t size = width * countRowsPerIter;
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(countRowsPerIter / threadsPerBlock.x + 1, width / threadsPerBlock.y + 1);

	double* newLon = new double[size];
	double* newLat = new double[size];
	double* dev_lon = 0;
	double* dev_lat = 0;

	float time;
	hipSetDevice(0);
	hipMalloc((void**)&dev_lon, size * sizeof(double));
	hipMalloc((void**)&dev_lat, size * sizeof(double));
	GPU_TIMER_START;
	for (int i = 0; i < countIter; i++)
	{
		double newYOrigin = yOrigin + i * yPixelSize * countRowsPerIter;

		applyTransformUtmToWgsCoordsGpu << <numBlocks, threadsPerBlock >> > (xOrigin, newYOrigin,
			xPixelSize, yPixelSize, countRowsPerIter, width, zone, southhemi, dev_lon, dev_lat);
		hipDeviceSynchronize();

		hipMemcpy(newLon, dev_lon, size * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(newLat, dev_lat, size * sizeof(double), hipMemcpyDeviceToHost);

		size_t countCoordsForCopy = i != countIter - 1 ? size :
			width * height - countRowsPerIter * width * i;
		for (int j = 0; j < countCoordsForCopy; j++)
		{
			lon[i * size + j] = newLon[j];
			lat[i * size + j] = newLat[j];
		}
	}
	GPU_TIMER_STOP(time);
	hipFree(dev_lon);
	hipFree(dev_lat);
	delete[] newLon;
	delete[] newLat;

	return (double)time;
}
