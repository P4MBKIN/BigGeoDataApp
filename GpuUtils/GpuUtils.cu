#include "hip/hip_runtime.h"
#include "GpuUtils.cuh"
#include "GpuFocalProcessing.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void addKernelGpu(double *res, const double *a, const double *b)
{
	int i = threadIdx.x;
	res[i] = a[i] + b[i];
}

void winGpu::testPlusGpu(const double* a, const double* b, double* res, size_t size)
{
	double* devA = 0;
	double* devB = 0;
	double* devRes = 0;
	hipSetDevice(0);
	hipMalloc((void**)&devRes, size * sizeof(double));
	hipMalloc((void**)&devA, size * sizeof(double));
	hipMalloc((void**)&devB, size * sizeof(double));
	hipMemcpy(devA, a, size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devB, b, size * sizeof(double), hipMemcpyHostToDevice);
	addKernelGpu << <1, (int)size >> > (devRes, devA, devB);
	hipDeviceSynchronize();
	hipMemcpy(res, devRes, size * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(devA);
	hipFree(devB);
	hipFree(devRes);
}

void winGpu::performFocalOpGpu(pixel* input, int height, int width, pixel* output, int type)
{
	winGpu::doFocalOpGpu(input, height, width, output, type);
}