#include "hip/hip_runtime.h"
#include "GpuUtils.cuh"
#include "GpuFocalProcessing.cuh"
#include "GpuProjectionProcessing.cuh"
#include "GpuTimer.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void addKernelGpu(double* res, const double* a, const double* b)
{
	int i = threadIdx.x;
	res[i] = a[i] + b[i];
}

double winGpu::testPlusGpu(const double* a, const double* b, double* res, size_t size)
{
	double* devA = 0;
	double* devB = 0;
	double* devRes = 0;
	hipSetDevice(0);
	hipMalloc((void**)&devRes, size * sizeof(double));
	hipMalloc((void**)&devA, size * sizeof(double));
	hipMalloc((void**)&devB, size * sizeof(double));
	hipMemcpy(devA, a, size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devB, b, size * sizeof(double), hipMemcpyHostToDevice);

	float time;
	GPU_TIMER_START;
	addKernelGpu << <1, (int)size >> > (devRes, devA, devB);
	hipDeviceSynchronize();
	GPU_TIMER_STOP(time);

	hipMemcpy(res, devRes, size * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(devA);
	hipFree(devB);
	hipFree(devRes);

	return (double)time;
}

double winGpu::performFocalOpGpu(pixel* input, int height, int width, pixel* output, std::vector<double> matrix)
{
	return winGpu::doFocalOpGpu(input, height, width, output, matrix);
}

double winGpu::performTransformUtmToWgsCoordsGpu(double xOrigin, double yOrigin, double xPixelSize, double yPixelSize,
	int height, int width, int zone, bool southhemi, double* lon, double* lat)
{
	return winGpu::doTransformUtmToWgsCoordsGpu(xOrigin, yOrigin, xPixelSize, yPixelSize,
		height, width, zone, southhemi, lon, lat);
}

double winGpu::performTransformWgsToUtmCoordsGpu(double xOrigin, double yOrigin, double xPixelSize, double yPixelSize,
	int height, int width, int zone, double* x, double* y)
{
	return winGpu::doTransformWgsToUtmCoordsGpu(xOrigin, yOrigin, xPixelSize, yPixelSize,
		height, width, zone, x, y);
}
