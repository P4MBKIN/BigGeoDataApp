#include "hip/hip_runtime.h"
#include "GpuFocalProcessing.cuh"
#include "GpuTimer.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cmath>

using namespace winGpu;

__global__ void applyFocalOpGpu(FocalRasterGpu rasterInput, FocalRasterGpu rasterOutput, FocalKernelGpu kernel, int rowIter)
{
	int h = blockDim.x * blockIdx.x + threadIdx.x + rowIter;
	int w = blockDim.y * blockIdx.y + threadIdx.y;
	if (rasterInput.height <= h || rasterInput.width <= w)
	{
		return;
	}

	if (rasterInput(h, w) == rasterInput.defaultValue)
	{
		rasterOutput(h, w) = rasterInput(h, w);
		return;
	}
	double sum = 0.0;
	for (int i = 0; i < kernel.sideSize; ++i)
	{
		for (int j = 0; j < kernel.sideSize; ++j)
		{
			pixel value = rasterInput(h + (i - kernel.midSize), w + (j - kernel.midSize));
			if (value == rasterInput.defaultValue)
			{
				rasterOutput(h, w) = rasterInput(h, w);
				return;
			}
			sum += kernel[i][j] * value;
		}
	}
	if (sum <= 0)
	{
		sum = 0.0;
	}
	rasterOutput(h, w) = (pixel)sum;
}

double winGpu::doFocalOpGpu(pixel* input, int height, int width, pixel* output, std::vector<double> matrix)
{
	// ������� Rater ��� ������� ������
	FocalRasterGpu rasterInput;
	rasterInput.height = height;
	rasterInput.width = width;
	rasterInput.data = 0;

	// ������� Rater ��� �������� ������
	FocalRasterGpu rasterOutput;
	rasterOutput.height = height;
	rasterOutput.width = width;
	rasterOutput.data = 0;

	// ������� Kernel ��� ���������� ������� �������
	FocalKernelGpu kernelTemp;
	kernelTemp.sideSize = (int)std::sqrt(matrix.size());
	kernelTemp.ker = matrix.data();
	kernelTemp.midSize = kernelTemp.sideSize / 2;

	FocalKernelGpu kernel;
	kernel.sideSize = kernelTemp.sideSize;
	kernel.midSize = kernelTemp.midSize;
	kernel.ker = 0;

	hipSetDevice(0);
	hipMalloc((void**)&rasterInput.data, rasterInput.size());
	hipMalloc((void**)&rasterOutput.data, rasterOutput.size());
	hipMalloc((void**)&kernel.ker, kernel.size());
	hipMemcpy(rasterInput.data, input, rasterInput.size(), hipMemcpyHostToDevice);
	hipMemcpy(kernel.ker, kernelTemp.ker, kernel.size(), hipMemcpyHostToDevice);

	const size_t maxAvaliableCoords = 8000000;
	int countRowsPerIter = maxAvaliableCoords / width;
	int countIter = height / countRowsPerIter + 1;
	const size_t size = width * countRowsPerIter;
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(countRowsPerIter / threadsPerBlock.x + 1, width / threadsPerBlock.y + 1);

	float time;
	GPU_TIMER_START;
	for (int i = 0; i < countIter; i++)
	{
		int rowIter = i * countRowsPerIter;
		applyFocalOpGpu << <numBlocks, threadsPerBlock >> > (rasterInput, rasterOutput, kernel, rowIter);
		hipDeviceSynchronize();
		int k = 5;
	}
	GPU_TIMER_STOP(time);

	hipMemcpy(output, rasterOutput.data, rasterOutput.size(), hipMemcpyDeviceToHost);

	hipFree(rasterInput.data);
	hipFree(rasterOutput.data);
	hipFree(kernel.ker);

	return (double)time;
}
