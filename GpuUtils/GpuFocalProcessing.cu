#include "hip/hip_runtime.h"
#include "GpuFocalProcessing.cuh"
#include "GpuTimer.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

using namespace winGpu;

__global__ void applyFocalOpGpu(FocalRasterGpu rasterInput, FocalRasterGpu rasterOutput, FocalKernelGpu kernel, int rowIter)
{
	int h = blockDim.x * blockIdx.x + threadIdx.x + rowIter;
	int w = blockDim.y * blockIdx.y + threadIdx.y;
	if (rasterInput.height <= h || rasterInput.width <= w)
	{
		return;
	}

	if (rasterInput(h, w) == rasterInput.defaultValue)
	{
		rasterOutput(h, w) = rasterInput(h, w);
		return;
	}
	double sum = 0.0;
	for (int i = 0; i < kernel.sideSize; ++i)
	{
		for (int j = 0; j < kernel.sideSize; ++j)
		{
			pixel value = rasterInput(h + (i - kernel.midSize), w + (j - kernel.midSize));
			if (value == rasterInput.defaultValue)
			{
				rasterOutput(h, w) = rasterInput(h, w);
				return;
			}
			sum += kernel[i][j] * value;
		}
	}
	if (sum <= 0)
	{
		sum = 0.0;
	}
	rasterOutput(h, w) = (pixel)sum;
}

double winGpu::doFocalOpGpu(pixel* input, int height, int width, pixel* output, int type)
{
	// ������� Rater ��� ������� ������
	FocalRasterGpu rasterInput;
	rasterInput.height = height;
	rasterInput.width = width;
	rasterInput.data = 0;

	// ������� Rater ��� �������� ������
	FocalRasterGpu rasterOutput;
	rasterOutput.height = height;
	rasterOutput.width = width;
	rasterOutput.data = 0;

	// ������� Kernel ��� ���������� ������� �������
	FocalKernelGpu kernelTemp;
	switch (type)
	{
	case FocalOpTypeGpu::BoxBlur3:
	{
		kernelTemp.sideSize = 3;
		double mas[] = GPU_BOX_BLUR_3;
		kernelTemp.ker = mas;
		break;
	}
	case FocalOpTypeGpu::BoxBlur5:
	{
		kernelTemp.sideSize = 5;
		double mas[] = GPU_BOX_BLUR_5;
		kernelTemp.ker = mas;
		break;
	}
	case FocalOpTypeGpu::BoxBlur7:
	{
		kernelTemp.sideSize = 7;
		double mas[] = GPU_BOX_BLUR_7;
		kernelTemp.ker = mas;
		break;
	}
	case FocalOpTypeGpu::GaussianBlur3:
	{
		kernelTemp.sideSize = 3;
		double mas[] = GPU_GAUSSIAN_BLUR_3;
		kernelTemp.ker = mas;
		break;
	}
	case FocalOpTypeGpu::GaussianBlur5:
	{
		kernelTemp.sideSize = 5;
		double mas[] = GPU_GAUSSIAN_BLUR_5;
		kernelTemp.ker = mas;
		break;
	}
	case FocalOpTypeGpu::EdgeDetection3_1:
	{
		kernelTemp.sideSize = 3;
		double mas[] = GPU_EDGE_DETECTION_3_1;
		kernelTemp.ker = mas;
		break;
	}
	case FocalOpTypeGpu::EdgeDetection3_2:
	{
		kernelTemp.sideSize = 3;
		double mas[] = GPU_EDGE_DETECTION_3_2;
		kernelTemp.ker = mas;
		break;
	}
	case FocalOpTypeGpu::EdgeDetection3_3:
	{
		kernelTemp.sideSize = 3;
		double mas[] = GPU_EDGE_DETECTION_3_3;
		kernelTemp.ker = mas;
		break;
	}
	case FocalOpTypeGpu::Sharpen3:
	{
		kernelTemp.sideSize = 3;
		double mas[] = GPU_SHARPEN_3;
		kernelTemp.ker = mas;
		break;
	}
	case FocalOpTypeGpu::UnsharpMasking5:
	{
		kernelTemp.sideSize = 3;
		double mas[] = GPU_UNSHARP_MASKING_5;
		kernelTemp.ker = mas;
		break;
	}
	default:
		break;
	}
	kernelTemp.midSize = kernelTemp.sideSize / 2;
	FocalKernelGpu kernel;
	kernel.sideSize = kernelTemp.sideSize;
	kernel.midSize = kernelTemp.midSize;
	kernel.ker = 0;

	hipSetDevice(0);
	hipMalloc((void**)&rasterInput.data, rasterInput.size());
	hipMalloc((void**)&rasterOutput.data, rasterOutput.size());
	hipMalloc((void**)&kernel.ker, kernel.size());
	hipMemcpy(rasterInput.data, input, rasterInput.size(), hipMemcpyHostToDevice);
	hipMemcpy(kernel.ker, kernelTemp.ker, kernel.size(), hipMemcpyHostToDevice);

	const size_t maxAvaliableCoords = 8000000;
	int countRowsPerIter = maxAvaliableCoords / width;
	int countIter = height / countRowsPerIter + 1;
	const size_t size = width * countRowsPerIter;
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(countRowsPerIter / threadsPerBlock.x + 1, width / threadsPerBlock.y + 1);

	float time;
	GPU_TIMER_START;
	for (int i = 0; i < countIter; i++)
	{
		int rowIter = i * countRowsPerIter;
		applyFocalOpGpu << <numBlocks, threadsPerBlock >> > (rasterInput, rasterOutput, kernel, rowIter);
		hipDeviceSynchronize();
		int k = 5;
	}
	GPU_TIMER_STOP(time);

	hipMemcpy(output, rasterOutput.data, rasterOutput.size(), hipMemcpyDeviceToHost);

	hipFree(rasterInput.data);
	hipFree(rasterOutput.data);
	hipFree(kernel.ker);

	return (double)time;
}
